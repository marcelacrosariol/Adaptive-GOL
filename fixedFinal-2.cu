#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <omp.h>

#define RANDVAL 1984
#define BLOCK_SIZE 16
#define DIM     4 // Linear dimension of our grid - not counting ghost cells

// Create an array that stores the number of rows of the subGrid in each device
__host__ void RowCount(int devCount,int *subGridSize){
    for (int i = 0; i < devCount; i++){
        if (DIM % devCount == 0)
            subGridSize[i] = DIM / devCount;
        else{
            if (i == 0) // If it is not possible to divide the rows equally between the devices, the first GPU will receive more rows than the others
                subGridSize[i] = ((int) DIM / devCount) + DIM % devCount;
            else
                subGridSize[i] = (int) DIM / devCount;
        }
    }
}

// Return the number of rows that exist in the main grid before the first row of the actual subgrid
__host__ int rowsBefore(int device, int *subGridSize){
    if (device == 0)
        return 0;
    else if (device ==1)
        return subGridSize[0];
    else
        return subGridSize[device-1] + rowsBefore(device-1, subGridSize);
}

__host__ int getLastRow(int device, int *subGridSize){
    if (device == 0)
        return subGridSize[0];
    else
        return subGridSize[device] + getLastRow(device-1, subGridSize);
}

__host__ void buildHaloCells(int *h_grid){
    // Copy halo rows
    for (int j = 1; j<= DIM; j++){

        // Copy first real row to last halo row
        h_grid[(DIM+1) * (DIM+2) + j] = h_grid[1 * (DIM+2) + j];

        // Copy last real row to first halo row
        h_grid[j] = h_grid[DIM * (DIM+2) + j];
    }

    // Copy halo columns
    for (int i = 0; i<= DIM+1; i++){

        // Copy first real column to last halo column
        h_grid[i * (DIM+2) + (DIM+1)] = h_grid[i* (DIM+2) + 1];

        // Copy last real column to first halo column
        h_grid[i * (DIM+2)] = h_grid[i * (DIM+2) + DIM];
    }
}

__host__ void buildSubGrid(int *h_grid, int *h_subGrid, int firstRow, int lastRow, int d){
    for(int i = firstRow-1; i<= lastRow+1; i++){
        for(int j = 0; j<= DIM+1; j++){
            h_subGrid[i * (DIM+2) + j] = h_grid[i * (DIM+2) + j];
        }
    }
}


__global__ void ghostRows(int *grid){
    // We want id ∈ [1,DIM]
    int id = blockDim.x * blockIdx.x + threadIdx.x + 1;

    if (id <= DIM){
        //Copy first real row to bottom ghost row
        grid[(DIM+2)*(DIM+1)+id] = grid[(DIM+2)+id];

        //Copy last real row to top ghost row
        grid[id] = grid[(DIM+2)*DIM + id];
    }
}

__global__ void ghostCols(int *grid){
    // We want id ∈ [0,DIM+1]
    int id = blockDim.x * blockIdx.x + threadIdx.x;

    if (id <= DIM+1){
        //Copy first real column to right most ghost column
        grid[id*(DIM+2)+DIM+1] = grid[id*(DIM+2)+1];

        //Copy last real column to left most ghost column
        grid[id*(DIM+2)] = grid[id*(DIM+2) + DIM];
    }
}

__global__ void GOL(int *grid, int *newGrid, int firstRow){
    // We want id ∈ [1,DIM]
    int iy = blockDim.y * blockIdx.y + threadIdx.y  + firstRow;
    int ix = blockDim.x * blockIdx.x + threadIdx.x + 1;
    int id = iy * (DIM+2) + ix;


    int numNeighbors;

    if (iy <= DIM && ix <= DIM) {

        // Get the number of neighbors for a given grid point
        numNeighbors = grid[id+(DIM+2)] + grid[id-(DIM+2)] //upper lower
        + grid[id+1] + grid[id-1]             //right left
        + grid[id+(DIM+3)] + grid[id-(DIM+3)] //diagonals
        + grid[id-(DIM+1)] + grid[id+(DIM+1)];

        int cell = grid[id];

        //printf("firstrow: %d ID: %d Grid[%d]: %d cell: %d Neighboors: %d \n", firstRow, id, id, grid[id], cell ,numNeighbors);

        // Here we have explicitly all of the game rules
        if (cell == 1 && numNeighbors < 2)
            newGrid[id] = 0;
        else if (cell == 1 && (numNeighbors == 2 || numNeighbors == 3))
            newGrid[id] = 1;
        else if (cell == 1 && numNeighbors > 3)
            newGrid[id] = 0;
        else if (cell == 0 && numNeighbors == 3)
            newGrid[id] = 1;
        else
            newGrid[id] = cell;
    }
}


int main(int argc, char* argv[]){

    int devCount;
    hipGetDeviceCount(&devCount); // Get the number of devices that the system have
    printf("There are %d devices \n", devCount);
    // If there is no GPU, it is not possible to run this version of Game of Life
    if (devCount == 0){
        printf("There are no devices in this machine!");
        return 0; // if there is no GPU, then break the code
    }
    
    int i, j, iter;
    int alive = 0, lim = DIM;
    int *h_grid;
    size_t gridBytes;
    
    gridBytes = sizeof(int)*(DIM+2)*(DIM+2); // 2 added for periodic boundary condition ghost cells
    // Alocate memory for host grid
    h_grid = (int*)malloc(gridBytes);
    
    srand(RANDVAL);
    // Assign random value to cells of the grid
    #pragma omp parallel for private(i,j)
    for(i = 1; i<=DIM; i++) {
        for(j = 1; j<=DIM; j++) {
           h_grid[i*(DIM+2)+j] = rand() % 2;
        }
    } // End of pragma


printf("\n");
for(int i = 1; i <= DIM; i++){
for(int j = 1; j <= DIM; j++){
printf("%d  ", h_grid[i*(DIM+2)+j]);
}
printf("\n");
}


    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE,1);
    int  linGrid = (int)ceil(DIM/(float)BLOCK_SIZE);
    dim3 gridSize(linGrid,linGrid,1);

    dim3 cpyBlockSize(BLOCK_SIZE,1,1);
    dim3 cpyGridRowsGridSize((int)ceil(DIM/(float)cpyBlockSize.x),1,1);
    dim3 cpyGridColsGridSize((int)ceil((DIM+2)/(float)cpyBlockSize.x),1,1);
    
    if (devCount == 1){
        int *d_grid, *d_newGrid, *d_tmpGrid;
        
        // Allocate device grids - if there is more than 1 thread, It'll allocate memory in each device
        hipMalloc(&d_grid, gridBytes);
        hipMalloc(&d_newGrid, gridBytes);
        
        // Copy over initial game grid (Dim-1 threads)
        hipMemcpy(d_grid, h_grid, gridBytes, hipMemcpyHostToDevice);

        for (iter = 0; iter < lim; iter ++){

            ghostRows<<<cpyGridRowsGridSize, cpyBlockSize>>>(d_grid);
            ghostCols<<<cpyGridColsGridSize, cpyBlockSize>>>(d_grid);
            GOL<<<gridSize, blockSize>>>(d_grid, d_newGrid,1);

            // Swap our grids and iterate again
            d_tmpGrid = d_grid;
            d_grid = d_newGrid;
            d_newGrid = d_tmpGrid;
        }

        // Copy back results and sum
        hipMemcpy(h_grid, d_grid, gridBytes, hipMemcpyDeviceToHost);
        
        // calculate the total of cells alive after the iteractions
        //#pragma omp parallel for private(i,j,alive)
        for (i = 1; i <= DIM; i++){
            for ( j =1 ; j <= DIM; j++){
                alive += h_grid[i*(DIM+2)+j];
            }
        }  // end of prama
        
        printf("There are %d cells alive after the last iteration\n", alive);
        
        // Release memory
        hipFree(d_grid);
        hipFree(d_newGrid);
        free(h_grid);
        
        return 1;
    }
    
    if (devCount > 1){

        int *h_SubGridSize;
        int CPUthreadId, currentDevice, firstRow, lastRow;
        int *h_subGrid;
        int *d_subGrid, *d_tempSub;
        size_t subBytes;

        h_SubGridSize = (int*)malloc(sizeof(int)*devCount); // Allocate memory for the subGridSize, which stores the number of elements in each subGrids
        RowCount(devCount, h_SubGridSize); // Calculate the size of the subgrid in each GPU

        buildHaloCells(h_grid);

        printf("\n");
        for(int i = 0; i <= DIM+1; i++){
            for(int j = 0; j <= DIM+1; j++){
                printf("%d  ", h_grid[i*(DIM+2)+j]);
            }
        printf("\n");
        }

        omp_set_num_threads(devCount);

        for (iter = 0; iter < lim; iter ++){

            #pragma omp parallel
            {
                #pragma omp ordered for private(currentDevice, CPUthreadId, h_subGrid, subBytes, firstRow, lastRow)
                {
                for (int device = 0; device < devCount; device++){
                CPUthreadId = omp_get_thread_num(); // Get the id of the actual thread

                currentDevice = CPUthreadId;
                hipSetDevice(device); // Set device to be used

                subBytes = sizeof(int)* (DIM+2) * (DIM+2);  //(h_SubGridSize[currentDevice]+2) * (DIM+2);  number of rows + 2 halo/ghost rows + 2 halo/ghost columns
                h_subGrid = (int*)malloc(subBytes); //allocate memory for the subGrid

                // Allocate device grids - if there is more than 1 thread, It'll allocate memory in each device
                hipMalloc(&d_subGrid, subBytes);
                hipMalloc(&d_tempSub, subBytes);

                // Calculates the first row of the submatrix in the main matrix  - Does not count the ghost rows
                firstRow = rowsBefore(currentDevice, h_SubGridSize) + 1;
                // Calculates the last row of the submatrix in the main matrix
                lastRow = getLastRow(currentDevice, h_SubGridSize);

                buildSubGrid(h_grid, h_subGrid, firstRow, lastRow, currentDevice);

                    printf("iter : %d \n", iter);
                    printf("\n printing subgrid gpu %d  \n", currentDevice);
                    printf("firstrow: %d lastRow:%d \n", firstRow, lastRow);
                    for(int i = firstRow-1; i <= lastRow+1; i++){
                        for(int j = 0; j <= DIM+1; j++){
                            printf("%d  ", h_subGrid[i*(DIM+2)+j]);
                        }
                        printf("\n");
                    }


                hipMemcpy(d_subGrid, h_subGrid, subBytes, hipMemcpyHostToDevice);

                // call GOL function and the new values will go to the d_tempSub grid
                GOL<<<gridSize, blockSize>>>(d_subGrid, d_tempSub, firstRow);

                free(h_subGrid);
                h_subGrid = (int*)malloc(subBytes); //allocate memory for the subGrid

                hipMemcpy(h_subGrid, d_tempSub, subBytes, hipMemcpyDeviceToHost);

               for(int i = firstRow; i <= lastRow; i++){
                    for(int j = 1; j <= DIM; j++){
                        h_grid[i*(DIM+2)+j] = h_subGrid[i*(DIM+2)+j];
                    }
                }
}}} // End pragma
        } // End iteration


        printf("\n___\n\n");
        for(int i = 1; i<=DIM; i++) {
            for(int j = 1; j<=DIM; j++) {
                printf("%d  ", h_grid[i*(DIM+2)+j]);
            }
            printf("\n");
        }


        for (int i = 1; i <= DIM; i++){
            for (int j =1 ; j <= DIM; j++){
                alive += h_grid[i*(DIM+2)+j];
            }
        }  // end of prama

        printf("There are %d cells alive after the last iteration\n", alive);

        // Release memory
        free(h_grid);
        hipFree(d_tempSub);

        return 1;
    }
}
    